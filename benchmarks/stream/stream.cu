#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <chrono>

template <typename T> int calculateOptimalBlocks(T kernel, int threadsPerBlock);

// olmalloc_mode determines which allocator we use
// 0 is malloc (hmm only!) (default)
// 1 is hipMallocManaged
// 2 is malloc with hipMemAdvise
// 3 is hipMallocManaged with hipMemAdvise
// 4 is malloc with hipMemAdvise and SetAccessBy current GPU
// 5 is hipMallocManaged with hipMemAdvise and SetAccessBy current GPU
static int olmalloc_mode = 0;
void* olmalloc(size_t bytes)
{
    void* foo = nullptr;
    int currentDevice = 0;
    hipGetDevice(&currentDevice);  // Get ID of current GPU

    switch (olmalloc_mode)
    {
        case 0:
            foo = malloc(bytes);
            break;
        case 1:
            hipMallocManaged(&foo, bytes);
            break;
        case 2:
        case 4:
            foo = malloc(bytes);
            hipMemAdvise(foo, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
            if (olmalloc_mode == 4) 
                hipMemAdvise(foo, bytes, hipMemAdviseSetAccessedBy, currentDevice);
            break;
        case 3:
        case 5:
            hipMallocManaged(&foo, bytes);
            hipMemAdvise(foo, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
            if (olmalloc_mode == 5) 
                hipMemAdvise(foo, bytes, hipMemAdviseSetAccessedBy, currentDevice);
            break;
        default:
            fprintf(stderr, "invalid olmalloc mode %d\n", olmalloc_mode);
            exit(1);
    }
    if (foo == nullptr)
    {
        fprintf(stderr, "olmalloc failed\n");
        exit(1);
    }
    return foo;
}

__global__ void warmup()
{
    return;
}

__global__ void triad(int* a, int* b, int* c, int scalar, size_t n) 
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        a[i] = b[i] + scalar * c[i];
    }
}

#define TPB 256
int main(int argc, char* argv[]) {
    size_t N;
    int scalar = 3;
    if (argc != 2 && argc != 3)
    {
        std::cerr << "Usage: " << argv[0] << " <size_t> [alloc_mode]\n";
        return 1;
    }
    try
    {
        N = std::stoul(argv[1]);
        if (argc == 3)
        {
            olmalloc_mode = std::stoi(argv[2]);
        }
    }
    catch (const std::invalid_argument& e) 
    {
        std::cerr << "Invalid argument: the input is not an unsigned integer.\n";
        return 2;
    }
    catch (const std::out_of_range& e) 
    {
        std::cerr << "Invalid argument: the input is out of range for a size_t.\n";
        return 3;
    }

    const int blocks = calculateOptimalBlocks(triad, TPB);
    int* a = (int*) olmalloc(sizeof(int) * N);
    int* b = (int*) olmalloc(sizeof(int) * N);
    int* c = (int*) olmalloc(sizeof(int) * N);

    printf("Allocating %lu bytes\n", sizeof(int) * N);
    printf("Allocating %lf gigabytes\n", sizeof(int) * N / 1e9);
    printf("Kernel Config: %d, %d\n", blocks, TPB);

    for (size_t i = 0; i < N; ++i)
    {
        a[i] = 0.0;
        b[i] = 2.0;
        c[i] = 1.0;
    }

    warmup<<<1, 1>>>();
    hipDeviceSynchronize();

    printf("launching triad with %d * %d = %d threads for %lu elements\n", blocks, TPB, blocks*TPB, N);

    auto start = std::chrono::high_resolution_clock::now();
    triad<<<blocks, TPB>>>(a, b, c, scalar, N);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);

    std::cerr << duration.count()/1e9 << std::endl;
    std::cout << "runtime: " << duration.count()/1e9 << " seconds" << std::endl;

    for (int i = 0; i < 10; ++i)
    {
        printf("a[%d] = %d\n", i, a[i]);
    }
    for (size_t i = 0; i < N; ++i)
    {
        if (a[i] != b[i] + scalar * c[i])
        {
            printf("i=%lu assert(%d == %d + %d * %d);\n", i, a[i], b[i], scalar, c[i]);
            assert(a[i] == b[i] + scalar * c[i]);
        }
    }
}

template <typename T>
int calculateOptimalBlocks(T kernel, int threadsPerBlock) {
    int device;
    hipDeviceProp_t props;

    // Get the device
    hipGetDevice(&device);

    // Get the device properties
    hipGetDeviceProperties(&props, device);

    // Get kernel attributes
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(kernel));

    // The maximum number of blocks is determined by the device compute capability
    int maxBlocksPerSM;
    switch (props.major) 
    {
        case 1:  // Compute capability 1.x
            maxBlocksPerSM = 8;
            break;
        case 2:  // Compute capability 2.x
            maxBlocksPerSM = 8;
            break;
        case 3:  // Compute capability 3.x
            maxBlocksPerSM = 16;
            break;
        case 5:  // Compute capability 5.x
            maxBlocksPerSM = 32;
            break;
        case 6:  // Compute capability 6.x
            if (props.minor == 0) maxBlocksPerSM = 40;
            else maxBlocksPerSM = 32;
            break;
        case 7:  // Compute capability 7.x
            maxBlocksPerSM = 16;
            break;
        default:  // Compute capability 8.x or above
            maxBlocksPerSM = 16;
            break;
    }

    // Adjust for the maximum number of threads per multiprocessor
    int maxBlocksByThreads = props.maxThreadsPerMultiProcessor / threadsPerBlock;
    maxBlocksPerSM = min(maxBlocksPerSM, maxBlocksByThreads);

    // Adjust for the amount of shared memory used by the kernel
    if (attr.sharedSizeBytes > 0) {
        int maxBlocksBySharedMem = props.sharedMemPerMultiprocessor / attr.sharedSizeBytes;
        maxBlocksPerSM = min(maxBlocksPerSM, maxBlocksBySharedMem);
    }

    // Adjust for the number of registers used by the kernel
    if (attr.numRegs > 0) {
        int maxBlocksByRegs = props.regsPerMultiprocessor / (attr.numRegs * threadsPerBlock);
        maxBlocksPerSM = min(maxBlocksPerSM, maxBlocksByRegs);
    }

    // Calculate the total number of blocks that can be resident on the GPU simultaneously
    int totalBlocks = maxBlocksPerSM * props.multiProcessorCount;

    return totalBlocks;
}
