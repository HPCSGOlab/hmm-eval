#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "mmio.h"

#define BlockDim 1024
#define MAX_NUM_THREADS_PER_BLOCK 1024
#define ITER 3

template <typename T>
__global__ void spmv_csr_scalar_kernel(T * d_val, T * d_vector, int * d_cols, int * d_ptr, int N, T * d_out) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
		T t = 0;
		int start = d_ptr[i];
		int end = d_ptr[i + 1];
		for (int j = start; j < end; j++) {
			int col = d_cols[j];
			t += d_val[j] * d_vector[col];
		}
		d_out[i] = t;
	}
}

template <typename T, int THREADS_PER_VECTOR, int MAX_NUM_VECTORS_PER_BLOCK>
__global__ void spmv_light_kernel(int* cudaRowCounter, int* d_ptr, int* d_cols,T* d_val, T* d_vector, T* d_out,int N) {
	int i;
	T sum;
	int row;
	int rowStart, rowEnd;
	int laneId = threadIdx.x % THREADS_PER_VECTOR; //lane index in the vector
	int vectorId = threadIdx.x / THREADS_PER_VECTOR; //vector index in the thread block
	int warpLaneId = threadIdx.x & 31;	//lane index in the warp
	int warpVectorId = warpLaneId / THREADS_PER_VECTOR;	//vector index in the warp

	__shared__ volatile int space[MAX_NUM_VECTORS_PER_BLOCK][2];

	// Get the row index
	if (warpLaneId == 0) {
		row = atomicAdd(cudaRowCounter, 32 / THREADS_PER_VECTOR);
	}
	// Broadcast the value to other threads in the same warp and compute the row index of each vector
		row = __shfl_sync(0xffffffff,row, 0) + warpVectorId;
	
	while (row < N) {

		// Use two threads to fetch the row offset
		if (laneId < 2) {
			space[vectorId][laneId] = d_ptr[row + laneId];
		}
		rowStart = space[vectorId][0];
		rowEnd = space[vectorId][1];

		sum = 0;
		// Compute dot product
		if (THREADS_PER_VECTOR == 32) {

			// Ensure aligned memory access
			i = rowStart - (rowStart & (THREADS_PER_VECTOR - 1)) + laneId;

			// Process the unaligned part
			if (i >= rowStart && i < rowEnd) {
				sum += d_val[i] * d_vector[d_cols[i]];
			}

				// Process the aligned part
			for (i += THREADS_PER_VECTOR; i < rowEnd; i += THREADS_PER_VECTOR) {
				sum += d_val[i] * d_vector[d_cols[i]];
			}
		} else {
			for (i = rowStart + laneId; i < rowEnd; i +=
					THREADS_PER_VECTOR) {
				sum += d_val[i] * d_vector[d_cols[i]];
			}
		}
		// Intra-vector reduction
		for (i = THREADS_PER_VECTOR >> 1; i > 0; i >>= 1) {
				sum += __shfl_down_sync(0xffffffff,sum, i);
		}

		// Save the results
		if (laneId == 0) {
			d_out[row] = sum;
		}

		// Get a new row index
		if(warpLaneId == 0){
			row = atomicAdd(cudaRowCounter, 32 / THREADS_PER_VECTOR);
		}
		// Broadcast the row index to the other threads in the same warp and compute the row index of each vector
			row = __shfl_sync(0xffffffff,row, 0) + warpVectorId;

	}
}


template <typename T>
void spmv_light(MatrixInfo<T> * mat,T *vector,T *out)
{
    	T *d_vector,*d_val, *d_out;
    	int *d_cols, *d_ptr;
    	float time_taken;
    	double gflop = 2 * (double) mat->nz / 1e9;
    	int meanElementsPerRow = mat->nz/mat->M;
    	int *cudaRowCounter;

    	// Allocate memory on device
    	hipMalloc(&d_vector,mat->N*sizeof(T));
    	hipMalloc(&d_val,mat->nz*sizeof(T));
    	hipMalloc(&d_out,mat->M*sizeof(T));
    	hipMalloc(&d_cols,mat->nz*sizeof(int));
    	hipMalloc(&d_ptr,(mat->M+1)*sizeof(int));
    	hipMalloc(&cudaRowCounter, sizeof(int));

	// Copy from host memory to device memory
    	hipMemcpy(d_vector,vector,mat->N*sizeof(T),hipMemcpyHostToDevice);
    	hipMemcpy(d_val,mat->val,mat->nz*sizeof(T),hipMemcpyHostToDevice);
    	hipMemcpy(d_cols,mat->cIndex,mat->nz*sizeof(int),hipMemcpyHostToDevice);
    	hipMemcpy(d_ptr,mat->rIndex,(mat->M+1)*sizeof(int),hipMemcpyHostToDevice);
    	hipMemset(d_out, 0, mat->M*sizeof(T));
    	hipMemset(cudaRowCounter, 0, sizeof(int));

	// Choose the vector size depending on the NNZ/Row, run the kernel and time it
	if (0) {
		printf("HERE\n");
		if (meanElementsPerRow <= 2) {
			for (int i = 0; i < ITER; i++) {
				spmv_light_kernel<T, 2, MAX_NUM_THREADS_PER_BLOCK / 2><<<ceil(mat->M/(float)BlockDim), BlockDim>>>(
					cudaRowCounter, d_ptr, d_cols,d_val,d_vector,d_out,mat->M);
				hipMemset(cudaRowCounter, 0, sizeof(int));
			}
		} else if (meanElementsPerRow <= 4) {
			for (int i = 0; i < ITER; i++) {
				spmv_light_kernel<T, 4, MAX_NUM_THREADS_PER_BLOCK / 4><<<ceil(mat->M/(float)BlockDim), BlockDim>>>(
					cudaRowCounter, d_ptr, d_cols,d_val, d_vector, d_out,mat->M);
				hipMemset(cudaRowCounter, 0, sizeof(int));
			}
		} else if(meanElementsPerRow <= 64) {
			for (int i = 0; i < ITER; i++) {
				spmv_light_kernel<T, 8, MAX_NUM_THREADS_PER_BLOCK / 8><<<ceil(mat->M/(float)BlockDim), BlockDim>>>(
					cudaRowCounter,d_ptr,d_cols,d_val, d_vector, d_out,mat->M);
				hipMemset(cudaRowCounter, 0, sizeof(int));
			}
		} else {
			for (int i = 0; i < ITER; i++){
				spmv_light_kernel<T, 32, MAX_NUM_THREADS_PER_BLOCK / 32><<<ceil(mat->M/(float)BlockDim), BlockDim>>>(
					cudaRowCounter, d_ptr, d_cols,d_val, d_vector, d_out,mat->M);
				hipMemset(cudaRowCounter, 0, sizeof(int));
			}
		}
	}
	else {
		for (int i = 0; i < ITER; i++)
			spmv_csr_scalar_kernel<T><<<ceil(mat->M/(float)BlockDim), BlockDim>>>(d_val, d_vector, d_cols, d_ptr,mat->M,d_out);
	}

    		// Copy from device memory to host memory
    	hipMemcpy(out, d_out, mat->M*sizeof(T), hipMemcpyDeviceToHost);
    	
	// Free device memory	
		// Calculate and print out GFLOPs and GB/s
	
}
