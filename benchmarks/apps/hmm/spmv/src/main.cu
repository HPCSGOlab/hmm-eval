#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h> 
#include <hip/hip_runtime.h>
#include "mmio.h"
#include "spmv.cuh"
#include "utils.cu"

int main(int argc, char ** argv){
    	char *mFile; 
    	float *SPvec, *SPout;
        double *DPvec, *DPout;
   	double time_taken;
		
    	MatrixInfo<float> * SPnewMat = (MatrixInfo<float> *) malloc(sizeof(MatrixInfo<float>));
        MatrixInfo<double> * DPnewMat = (MatrixInfo<double> *) malloc(sizeof(MatrixInfo<double>));
    	mFile = argv[1];
	//printf("Reading matrix from %s\n", mFile);
	
    	hipEvent_t start, stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);
	
	//capture the data movement
    	hipEventRecord(start);


	// Load matrices as both single and double precision
	MatrixInfo<float> * SPmatrix = read_file<float>(mFile);
        MatrixInfo<double> * DPmatrix = read_file<double>(mFile);
	if(SPmatrix == NULL){
		printf("Error regarding matrix file.");
		return 1;
	}
        else
        {
          	//printf("Number of rows is %d\n",SPmatrix->M);
      		//printf("Number of columns is %d\n",SPmatrix->N);
		//printf("Number of non zeros is %d\n",SPmatrix->nz);
    		//printf("Average number of non zeros per row is %d\n\n",SPmatrix->nz/SPmatrix->M);
        }


	double gflop = 2 * (double) DPmatrix->nz / 1e9;
    	float milliseconds = 0;


	// Change from COO format to CSR format
        //printf("Changing sparse matrix format to CSR...\n");
	SPnewMat = transferMat<float>(SPmatrix);
        DPnewMat = transferMat<double>(DPmatrix);
	
        convert2CSR<float>(SPnewMat);
        convert2CSR<double>(DPnewMat);

	// Create dense vector in both single and double precision
        SPvec = write_vector<float>(SPnewMat->N);
     	DPvec = write_vector<double>(DPnewMat->N);
 
	// Run kernels and print results
	//printf("\nSingle Precision Results\n\n");
	/*
        SPout = (float *)malloc(SPnewMat->M*sizeof(float));
        spmv_csr_scalar<float>(SPnewMat, SPvec, SPout);
        verify<float>(SPmatrix->nz,SPmatrix->M,SPmatrix->rIndex,SPmatrix->cIndex,SPmatrix->val,SPvec,SPout);
        free(SPout);

        SPout = (float *)malloc(SPnewMat->M*sizeof(float));
        spmv_csr_vector(SPnewMat, SPvec, SPout);
	verify<float>(SPmatrix->nz,SPmatrix->M,SPmatrix->rIndex,SPmatrix->cIndex,SPmatrix->val,SPvec,SPout);
        free(SPout);        

	SPout = (float *)malloc(SPnewMat->M*sizeof(float));
        spmv_csr_adaptive(SPnewMat, SPvec, SPout);
        verify<float>(SPmatrix->nz,SPmatrix->M,SPmatrix->rIndex,SPmatrix->cIndex,SPmatrix->val,SPvec,SPout);
        free(SPout);
        
        SPout = (float *)malloc(SPnewMat->M*sizeof(float));
        spmv_pcsr(SPnewMat,SPvec,SPout);
        verify<float>(SPmatrix->nz,SPmatrix->M,SPmatrix->rIndex,SPmatrix->cIndex,SPmatrix->val,SPvec,SPout);
        free(SPout);
*/

	/* SINGLE PERCISION
	SPout = (float *)malloc(SPnewMat->M*sizeof(float));
        spmv_light(SPnewMat,SPvec,SPout);
        verify<float>(SPmatrix->nz,SPmatrix->M,SPmatrix->rIndex,SPmatrix->cIndex,SPmatrix->val,SPvec,SPout);
        free(SPout);
	*/
/*
        SPout = (float *)malloc(SPnewMat->M*sizeof(float));
        
        start = clock();
        spmv_cpu<float>(SPnewMat->val, SPvec, SPnewMat->cIndex,SPnewMat->rIndex,SPnewMat->M,SPout);
        end = clock();
        
        time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
        printf("Time taken for %s is %lf\n", "SpMV by CPU CSR Algorithm", time_taken);
        verify(SPmatrix->nz,SPmatrix->M,SPmatrix->rIndex,SPmatrix->cIndex,SPmatrix->val,SPvec,SPout);      	
	free(SPout);
	*/

	//printf("\nDouble Precision Results\n\n");       
	/*
	DPout = (double *)malloc(SPnewMat->M*sizeof(double));
        spmv_csr_scalar<double>(DPnewMat, DPvec, DPout);
        verify<double>(DPmatrix->nz,DPmatrix->M,DPmatrix->rIndex,DPmatrix->cIndex,DPmatrix->val,DPvec,DPout);
        free(DPout);

	DPout = (double *)malloc(SPnewMat->M*sizeof(double));
        spmv_csr_vector<double>(DPnewMat, DPvec, DPout);
        verify<double>(DPmatrix->nz,DPmatrix->M,DPmatrix->rIndex,DPmatrix->cIndex,DPmatrix->val,DPvec,DPout);
        free(DPout);

	DPout = (double *)malloc(SPnewMat->M*sizeof(double));
        spmv_csr_adaptive<double>(DPnewMat, DPvec, DPout);
        verify<double>(DPmatrix->nz,DPmatrix->M,DPmatrix->rIndex,DPmatrix->cIndex,DPmatrix->val,DPvec,DPout);
        free(DPout);
	
	DPout = (double *)malloc(SPnewMat->M*sizeof(double));
        spmv_pcsr<double>(DPnewMat, DPvec, DPout);
        verify<double>(DPmatrix->nz,DPmatrix->M,DPmatrix->rIndex,DPmatrix->cIndex,DPmatrix->val,DPvec,DPout);
        free(DPout);
	*/

	DPout = (double *)malloc(SPnewMat->M*sizeof(double));
        spmv_light<double>(DPnewMat, DPvec, DPout);
        verify<double>(DPmatrix->nz,DPmatrix->M,DPmatrix->rIndex,DPmatrix->cIndex,DPmatrix->val,DPvec,DPout);
        free(DPout);

	hipEventRecord(stop);
    	hipEventSynchronize(stop);
    	hipEventElapsedTime(&milliseconds, start, stop);

	// Calculate and print out GFLOPs and GB/s
	double gbs = ((DPmatrix->N * sizeof(double)) + (DPmatrix->nz*sizeof(double)) + (DPmatrix->M*sizeof(int)) + (DPmatrix->nz*sizeof(int)) + (DPmatrix->M*sizeof(double))) / (milliseconds/ITER) / 1e6;
    	time_taken = (milliseconds/ITER)/1000.0;
    	//printf("Average time taken for %s is %f\n", "SpMV by GPU CSR LightSpMV Algorithm",time_taken);
    	//printf("Average GFLOP/s is %lf\n",gflop/time_taken);
	//printf("Average GB/s is %lf\n\n",gbs);
	//Type,Size(KB),s,GB/s
	double size = ((DPmatrix->N * sizeof(double)) + (DPmatrix->nz*sizeof(double)) + (DPmatrix->M*sizeof(int)) + (DPmatrix->nz*sizeof(int)) + (DPmatrix->M*sizeof(double))) * 1.0E-6;
	printf("GPU,%f,%f,%f\n", size, time_taken, (gflop/time_taken));

	/*
	DPout = (double *)malloc(SPnewMat->M*sizeof(double));
	start = clock();
        spmv_cpu<double>(DPnewMat->val, DPvec, DPnewMat->cIndex,DPnewMat->rIndex,DPnewMat->M,DPout);
	end = clock();
	
	time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
        printf("Time taken for %s is %lf\n", "SpMV by CPU CSR Algorithm", time_taken);
        verify<double>(DPmatrix->nz,DPmatrix->M,DPmatrix->rIndex,DPmatrix->cIndex,DPmatrix->val,DPvec,DPout);
        free(DPout); 
	*/

	// Free up memory
        freeMatrixInfo<float>(SPmatrix);
	freeMatrixInfo<float>(SPnewMat);
	free(SPvec);

	freeMatrixInfo<double>(DPmatrix);
        freeMatrixInfo<double>(DPnewMat);
        free(DPvec);
}
